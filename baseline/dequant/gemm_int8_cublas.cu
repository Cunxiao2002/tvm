#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>

int main()
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int m = 256, n = 8192, k = 8192;
    std::srand(std::time(0));

    std::vector<int8_t> h_A(m * k);
    std::vector<int8_t> h_B(k * n);
    std::vector<int32_t> h_C(m * n);

    for (int i = 0; i < m * k; ++i)
    {
        h_A[i] = static_cast<int8_t>(std::rand() % 127);
    }
    for (int i = 0; i < k * n; ++i)
    {
        h_B[i] = static_cast<int8_t>(std::rand() % 127);
    }

    int8_t *A, *B;
    int32_t *C;
    hipMalloc(&A, m * k * sizeof(int8_t));
    hipMalloc(&B, k * n * sizeof(int8_t));
    hipMalloc(&C, m * n * sizeof(int32_t));

    hipMemcpy(A, h_A.data(), m * k * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(B, h_B.data(), k * n * sizeof(int8_t), hipMemcpyHostToDevice);

    const int32_t alpha = 1;
    const int32_t beta = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int num_iters = 100;
    float total_time = 0.0f;

    for (int iter = 0; iter < num_iters; ++iter)
    {
        hipEventRecord(start, 0);

        hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                     m, n, k,
                     &alpha,
                     A, HIP_R_8I, k,
                     B, HIP_R_8I, k,
                     &beta,
                     C, HIP_R_32I, m,
                     HIPBLAS_COMPUTE_32I_PEDANTIC, HIPBLAS_GEMM_DEFAULT);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        total_time += milliseconds;
    }

    float avg_time = total_time / num_iters;
    float total_flops = 2.0 * m * n * k;
    std::cout << "Average hipblasGemmEx execution time over " << num_iters << " runs: " << avg_time << " ms" << std::endl;
    std::cout << "Average TFLOPS: " << total_flops / (avg_time * 1e9) << std::endl;

    hipMemcpy(h_C.data(), C, m * n * sizeof(int32_t), hipMemcpyDeviceToHost);

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipblasDestroy(handle);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
